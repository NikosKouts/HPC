/*
* This sample implements a separable convolution 
* of a 2D image with an arbitrary filter.
*/

#include <stdio.h>
#include <stdlib.h>
#include<hip/hip_runtime.h>
#include <time.h>

unsigned int filter_radius;

#define FILTER_LENGTH 	(2 * filter_radius + 1)
#define ABS(val)  	((val)<0.0 ? (-(val)) : (val))
#define ACCURACY  	0.00005F
#define maximum_dimensions 32
#define BLOCK_DIMENSION 32
// #define CPU

#define cudaCheckError() {                                                                        \
  hipError_t error=hipGetLastError();                                                           \
  if(error!=hipSuccess) {                                                                        \
    fprintf(stderr, "ERROR IN CUDA %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(error));   \
    hipDeviceReset();                                                                            \
    exit(EXIT_FAILURE);                                                                           \
  }                                                                                               \
}


// Constant Memory
__constant__ double c_Filter[8192]; //Maximum Constant Memory


////////////////////////////////////////////////////////////////////////////////
// Reference row convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionRowCPU(double *h_Dst, double *h_Src, double *h_Filter, int imageW, int imageH, int filterR) {
  int x, y, k;

  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      double sum = 0;
      
      for (k = -filterR; k <= filterR; k++) {
        int d = x + k;
        if(d >= 0 && d < imageW) {
          sum += h_Src[y * imageW + d] * h_Filter[filterR - k];
        }     

        h_Dst[y * imageW + x] = sum;
      }
    }
  }      
}

////////////////////////////////////////////////////////////////////////////////
// Reference column convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionColumnCPU(double *h_Dst, double *h_Src, double *h_Filter, int imageW, int imageH, int filterR) {
  int x, y, k;
  
  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      double sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = y + k;

        if(d >= 0 && d < imageH) {
          sum += h_Src[d * imageW + x] * h_Filter[filterR - k];
        }   
 
        h_Dst[y * imageW + x] = sum;
      }
    }
  }   
}

__global__ void convolutionRowGPU(double *d_Dst, double *d_Src, int i, int j, int image_dimensions, int tile_width, int filterR){
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  double sum = 0;
  int k;
  int d;


  // Padded Shared Memory
  extern __shared__ double s_src[];

  // Left Padding
  if(tx < filterR){
    s_src[ty * (BLOCK_DIMENSION + 2 * filterR) + tx] = d_Src[(y + filterR) * tile_width + x];
  }

  //Right Padding
  if(tx + filterR > BLOCK_DIMENSION - 1){
    s_src[ty * (BLOCK_DIMENSION + 2 * filterR) + tx + 2 * filterR] = d_Src[(y + filterR) * tile_width + x + 2 * filterR];
  }


  //Actual Value
  s_src[ty * (BLOCK_DIMENSION + 2 * filterR) + tx + filterR] = d_Src[(y + filterR) * tile_width + x + filterR];

  __syncthreads();


  //Actual Convolution
  for (k = -filterR; k <= filterR; k++) {
    d = tx + k;

    sum += s_src[ty * (BLOCK_DIMENSION + 2 * filterR) + d + filterR] * c_Filter[filterR - k];  
  }
  

  d_Dst[(y + i) * image_dimensions + x + j] = sum;
}


__global__ void convolutionColumnGPU(double *d_Dst, double *d_Src, int i, int j, int image_dimensions, int tile_width, int filterR){
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  double sum = 0;
  int k;
  int d;

  // Padded Shared Memory
  extern __shared__ double s_src[];
  
  
  if(ty < filterR){
    s_src[ty * BLOCK_DIMENSION + tx] = d_Src[y * tile_width + x + filterR];
  }


  if(ty + filterR > BLOCK_DIMENSION - 1){
    s_src[(ty + 2 * filterR) * BLOCK_DIMENSION + tx] = d_Src[(y + 2 * filterR) * tile_width + x + filterR];
  }
  
  s_src[(ty + filterR) * BLOCK_DIMENSION + tx] = d_Src[(y + filterR) * tile_width + x + filterR];

  __syncthreads();


  for (k = -filterR; k <= filterR; k++) {
    d = ty + k;
  
    sum += s_src[(d + filterR) * BLOCK_DIMENSION + tx] * c_Filter[filterR - k];
  }

  d_Dst[(y + i) * image_dimensions + x + j] = sum;  
}

void grid_init(dim3 *grid_dimensions, dim3 *block_dimensions, int imageW, int imageH){
  //Default Grid Values
  grid_dimensions->x = 1; 
  grid_dimensions->y = 1; 
  
  //Block Dimensions (0 - 32)
  block_dimensions->x = imageW > maximum_dimensions ? maximum_dimensions : imageW;
  block_dimensions->y = imageH > maximum_dimensions ? maximum_dimensions : imageH;


  //Calculate Grid's Horizontal Geometry
  if(imageW > maximum_dimensions){
    grid_dimensions->x = (imageW / maximum_dimensions); 
    if(imageW % maximum_dimensions != 0)
      grid_dimensions->x++;
  }
  
  //Calculate Grid's Vertical Geometry
  if(imageH > maximum_dimensions){
    grid_dimensions->y = (imageH / maximum_dimensions); 
    if(imageH % maximum_dimensions != 0)
      grid_dimensions->y++;
  }
}



// Function that performs a tiled row convolution and returns a new image that is located on device for further processing
double *row_tiling_convolution(double *h_input, int image_dimensions, dim3 block_dimensions, dim3 grid_dimensions, int tile_count){
  double **h_tiles, **d_tiles;
  int tile_data_width, tile_data_height;
  int tile_width, tile_height;
  int shared_memory_size_horizontal;
  double *m_buffer;  
  int tile = 0;
  hipStream_t *streams;
  hipError_t *results;

  int total_tiles = tile_count * tile_count;

  // Set tile's Dimensions
  tile_data_width = tile_data_height = image_dimensions / tile_count;
  tile_width = tile_height = tile_data_width + 2 * filter_radius;

  // Buffer memory on device
  hipMallocManaged((void **) &m_buffer, image_dimensions * image_dimensions * sizeof(double));
  if(!m_buffer)
    exit(EXIT_FAILURE);


  //Array of tiles to support async copies
  h_tiles = (double **) malloc(total_tiles * sizeof(double *));
  if(!h_tiles)
    exit(EXIT_FAILURE);
  
  // Allocate memory for each tile
  for(int i = 0; i < total_tiles; i++){
    h_tiles[i] = (double *) calloc(tile_height * tile_width, sizeof(double));
    if(!h_tiles[i])
      exit(EXIT_FAILURE);
  }

  // Create Streams
  streams = (hipStream_t *) malloc(total_tiles * sizeof(hipStream_t));
  if(!streams)
    exit(EXIT_FAILURE);

  //Create Results Array
  results = (hipError_t *) malloc(total_tiles * sizeof(hipError_t));
  if(!results)
    exit(EXIT_FAILURE);
  
  // Initialize Streams and Results
  for(int i = 0; i < total_tiles; i++){
    results[i] = hipStreamCreate(streams + i);
  }


  // Host array of pointers to device arrays
  d_tiles = (double **) malloc(total_tiles * sizeof(double *));
  if(!d_tiles)
    exit(EXIT_FAILURE);
  
  // Allocate memory for each device tile
  for(int i = 0; i < total_tiles; i++){
    hipMalloc((void **) &d_tiles[i], tile_height * tile_width * sizeof(double));
    if(!d_tiles[i])
      exit(EXIT_FAILURE);
  }


  //Shared memory size for row convolution
  shared_memory_size_horizontal = block_dimensions.y * (block_dimensions.x + 2 * filter_radius);

  //Handle tiling, memory copies, and allocating memory on streams 
  for(int i = 0; i < image_dimensions; i += tile_data_height){
    for(int j = 0; j < image_dimensions; j += tile_data_width){
      
      // Steps inside block
      for(int ii = 0; ii < tile_data_height; ii++){
        for(int jj = 0; jj < tile_data_width; jj++){

          // Set Top and Bottom Respectively
          if(i != 0 && ii < filter_radius){
            h_tiles[tile][ii  * tile_width + jj + filter_radius] = h_input[(i + ii - filter_radius) * image_dimensions + (j + jj)];
          }
          else if(i < image_dimensions - tile_data_height && ii >= tile_data_height - filter_radius){
            h_tiles[tile][(ii + 2 * filter_radius)  * tile_width + jj + filter_radius] = h_input[(i + ii + filter_radius) * image_dimensions + (j + jj)];
          }
          

          // Set Left and Right Respectively
          if(j != 0 && jj < filter_radius){
            h_tiles[tile][(ii + filter_radius) * tile_width + jj] = h_input[(i + ii) * image_dimensions + (j + jj - filter_radius)];
          }
          else if(j < image_dimensions - tile_data_width && jj >= tile_data_width - filter_radius){
            h_tiles[tile][(ii + filter_radius) * tile_width + jj + 2 * filter_radius] = h_input[(i + ii) * tile_data_width + (j + jj + filter_radius)];;
          }

          // Set Current Postition
          h_tiles[tile][(ii + filter_radius) * tile_width + (jj + filter_radius)] = h_input[(i + ii) * image_dimensions + (j + jj)]; 
        }
      }
      

      // Copy memory to device
      hipMemcpyAsync(d_tiles[tile], h_tiles[tile], tile_height * tile_width * sizeof(double), hipMemcpyHostToDevice, streams[tile]);


      // printf("\n----------------- [ Tile %d %d ] -----------------\n", i, j);
      // for(int row = 0; row < tile_height; row++){
      //   for(int col = 0; col < tile_width; col++){
      //     printf("%lf ", h_tiles[tile][row * tile_width + col]);
      //   }
      //   printf("\n");
      // }
      tile++;  
    }
  }


  // Convolution on seperate streams
  tile = 0;
  for(int i = 0; i < image_dimensions; i += tile_data_height){
    for(int j = 0; j < image_dimensions; j += tile_data_width){
       // Start GPU convolution 
      convolutionRowGPU<<<grid_dimensions, block_dimensions, shared_memory_size_horizontal * sizeof(double), streams[tile]>>>(m_buffer, d_tiles[tile], i, j, image_dimensions, tile_width, filter_radius);
      cudaCheckError();

      tile++;
    }
  }
  
  hipDeviceSynchronize();

  //Destroy Streams
  for(int i = 0; i < total_tiles; i++){
    results[i] = hipStreamDestroy(streams[i]);
  }

  return m_buffer;
}



// Function that performs a tiled row convolution and returns a new image that is located on device for further processing
double *column_tiling_convolution(double *h_input, int image_dimensions, dim3 block_dimensions, dim3 grid_dimensions, int tile_count){
  double **h_tiles, **d_tiles;
  int tile_data_width, tile_data_height;
  int tile_width, tile_height;
  int shared_memory_size_vertical;
  double *m_output_GPU;
  hipStream_t *streams;
  hipError_t *results;
  int tile = 0;

  int total_tiles = tile_count * tile_count;

  // Set tile's Dimensions
  tile_data_width = tile_data_height = image_dimensions / tile_count;
  tile_width = tile_height = tile_data_width + 2 * filter_radius;

  //Unified Host - Device Memory (On-Demand memcpy())
  hipMallocManaged(&m_output_GPU, image_dimensions * image_dimensions * sizeof(double));
  if(!m_output_GPU)
    exit(EXIT_FAILURE);


  //Array of tiles to support async copies
  h_tiles = (double **) malloc(total_tiles * sizeof(double *));
  if(!h_tiles)
    exit(EXIT_FAILURE);
  
  // Allocate memory for each tile
  for(int i = 0; i < total_tiles; i++){
    h_tiles[i] = (double *) calloc(tile_height * tile_width, sizeof(double));
    if(!h_tiles[i])
      exit(EXIT_FAILURE);
  }

  // Create Streams
  streams = (hipStream_t *) malloc(total_tiles * sizeof(hipStream_t));
  if(!streams)
    exit(EXIT_FAILURE);

  //Create Results Array
  results = (hipError_t *) malloc(total_tiles * sizeof(hipError_t));
  if(!results)
    exit(EXIT_FAILURE);
  
  // Initialize Streams and Results
  for(int i = 0; i < total_tiles; i++){
    results[i] = hipStreamCreate(streams + i);
  }


  // Host array of pointers to device arrays
  d_tiles = (double **) malloc(total_tiles * sizeof(double *));
  if(!d_tiles)
    exit(EXIT_FAILURE);
  
  // Allocate memory for each device tile
  for(int i = 0; i < total_tiles; i++){
    hipMalloc((void **) &d_tiles[i], tile_height * tile_width * sizeof(double));
    if(!d_tiles[i])
      exit(EXIT_FAILURE);
  }


  // Shared memory size for column convolution
  shared_memory_size_vertical = (block_dimensions.y + 2 * filter_radius) * block_dimensions.x;


  for(int i = 0; i < image_dimensions; i += tile_data_height){
    for(int j = 0; j < image_dimensions; j += tile_data_width){
      
      // Steps inside block
      for(int ii = 0; ii < tile_data_height; ii++){
        for(int jj = 0; jj < tile_data_width; jj++){

          // Set Top and Bottom Respectively
          if(i != 0 && ii < filter_radius){
            h_tiles[tile][ii  * tile_width + jj + filter_radius] = h_input[(i + ii - filter_radius) * image_dimensions + (j + jj)];
          }
          else if(i < image_dimensions - tile_data_height && ii >= tile_data_height - filter_radius){
            h_tiles[tile][(ii + 2 * filter_radius)  * tile_width + jj + filter_radius] = h_input[(i + ii + filter_radius) * image_dimensions + (j + jj)];
          }
          

          // Set Left and Right Respectively
          if(j != 0 && jj < filter_radius){
            h_tiles[tile][(ii + filter_radius) * tile_width + jj] = h_input[(i + ii) * image_dimensions + (j + jj - filter_radius)];
          }
          else if(j < image_dimensions - tile_data_width && jj >= tile_data_width - filter_radius){
            h_tiles[tile][(ii + filter_radius) * tile_width + jj + 2 * filter_radius] = h_input[(i + ii) * tile_data_width + (j + jj + filter_radius)];;
          }

          // Set Current Postition
          h_tiles[tile][(ii + filter_radius) * tile_width + (jj + filter_radius)] = h_input[(i + ii) * image_dimensions + (j + jj)]; 
        }
      }
      
       // Copy memory to device
      hipMemcpyAsync(d_tiles[tile], h_tiles[tile], tile_height * tile_width * sizeof(double), hipMemcpyHostToDevice, streams[tile]);

      cudaCheckError();     
      
      tile++; 
    }
  }

  // Convolution on seperate streams
  tile = 0;
  for(int i = 0; i < image_dimensions; i += tile_data_height){
    for(int j = 0; j < image_dimensions; j += tile_data_width){
      // Start GPU convolution 
      convolutionColumnGPU<<<grid_dimensions, block_dimensions, shared_memory_size_vertical * sizeof(double), streams[tile]>>>(m_output_GPU, d_tiles[tile], i, j, image_dimensions, tile_width, filter_radius);

      cudaCheckError();

      tile++;
    }
  }

  hipDeviceSynchronize();

  //Destroy Streams
  for(int i = 0; i < total_tiles; i++){
    results[i] = hipStreamDestroy(streams[i]);
  }


  return m_output_GPU;
}





////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
  double
    *h_Filter,
    *h_Input,
    *h_Buffer,
    *h_OutputCPU, 
    *m_OutputGPU,
    *m_buffer,
    accuracy = ACCURACY,
    h_time_used,
    d_memcpy_time_used,
    d_computation_time_used;
  dim3 
    block_dimensions,
    grid_dimensions; 
  int 
    imageW,
    imageH,
    tile_count,
    tile_data_width,
    tile_data_height;
  unsigned int i;
  clock_t h_start, h_end, d_start, d_end;


	printf("Enter filter radius : ");
	scanf("%d", &filter_radius);

  printf("Enter image size. Should be a power of two and greater than %d : ", FILTER_LENGTH);
  scanf("%d", &imageW);
  imageH = imageW;

  printf("Enter the amount of tiles to split image: ");
  scanf("%d", &tile_count);

  
  // Allocate memory on host
  h_Filter    = (double *) malloc(FILTER_LENGTH * sizeof(double));
  h_Input     = (double *) malloc(imageW * imageH * sizeof(double));
  h_Buffer    = (double *) malloc(imageW * imageH * sizeof(double));
  h_OutputCPU = (double *) malloc(imageW * imageH * sizeof(double));


  // Check malloc() for host
  if(!h_Filter || !h_Input || !h_Buffer || !h_OutputCPU)
    return -1;

  // Seed for Random Numbers
  srand(200);

  for (i = 0; i < FILTER_LENGTH; i++) {
    //h_Filter[i] = (double)(rand() % 16);
    h_Filter[i] = 1;
  }

  for (i = 0; i < imageW * imageH; i++) {
    //h_Input[i] = (double)rand() / ((double)RAND_MAX / 255) + (double)rand() / (double)RAND_MAX;
    h_Input[i] = 1;
  }

#ifdef CPU

  // Start CPU computation
  printf("\033[0;31mCPU computation...\033[0m\n");

  //Start calculate CPU time
  h_start = clock();
 
  convolutionRowCPU(h_Buffer, h_Input, h_Filter, imageW, imageH, filter_radius); // convolution kata grammes
  convolutionColumnCPU(h_OutputCPU, h_Buffer, h_Filter, imageW, imageH, filter_radius); // convolution kata sthles
 
  //End calculate CPU time
  h_end = clock();
  h_time_used = ((double) (h_end - h_start)) / CLOCKS_PER_SEC;

#endif

  //********************************************** GPU **********************************************// 
  
  // Set tile's Dimensions
  tile_data_width = tile_data_height = imageW / tile_count;
  

  //Initialize Grid
  grid_init(&grid_dimensions, &block_dimensions, tile_data_width, tile_data_height);
  printf("Grid(%d, %d), Block(%d, %d)\n", grid_dimensions.x, grid_dimensions.y, block_dimensions.x, block_dimensions.y);
  
  // Start GPU computation
  printf("\033[0;32mGPU computation...\033[0m\n");
  
  //Start calculate GPU memory copy time
  d_start = clock();

  // Copying filter from host to constant memory
  hipMemcpyToSymbol(HIP_SYMBOL(c_Filter), h_Filter, FILTER_LENGTH * sizeof(double), 0, hipMemcpyHostToDevice);
  
  d_end = clock();
  d_memcpy_time_used = ((double) (d_end - d_start)) / CLOCKS_PER_SEC;


  d_start = clock();

  

  //Convolution Calculation
  m_buffer = row_tiling_convolution(h_Input, imageW, block_dimensions, grid_dimensions, tile_count);
  m_OutputGPU = column_tiling_convolution(m_buffer, imageW,  block_dimensions, grid_dimensions, tile_count);
  
  cudaCheckError();

  //End calculate GPU time
  d_end = clock();
  d_computation_time_used = ((double) (d_end - d_start)) / CLOCKS_PER_SEC;



#ifdef CPU

  // Compare CPU and GPU results
  for(int i = 0; i < imageH * imageW; i++){
    if(ABS(h_OutputCPU[i] - m_OutputGPU[i]) > accuracy)
      accuracy = ABS(h_OutputCPU[i] - m_OutputGPU[i]);
  }

  printf("Comparing CPU and GPU with Accuracy %lf\n", accuracy);
  printf("\033[0;31mCPU Computation = %lf\033[0m\n", h_time_used);

#endif

  printf("\033[0;32mGPU Time = Memory Copy(%lf) + Computation(%lf) = %lf\033[0m\n", d_memcpy_time_used, d_computation_time_used, d_memcpy_time_used + d_computation_time_used);
  
  
  // Free all the allocated host memory ja
  free(h_OutputCPU);
  free(h_Buffer);
  free(h_Filter);
  free(h_Input);
  
  // Free all the allocated device memory
  hipFree(m_buffer);

  // Free all the allocated unified memory 
  hipFree(m_OutputGPU);
  

  // Do a device reset just in case...
  hipDeviceReset();
  
  return 0;
}
